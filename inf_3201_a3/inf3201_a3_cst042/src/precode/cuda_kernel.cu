
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void decipher(unsigned int, unsigned int*, unsigned int const*);


__global__ void decrypt_bytes(unsigned int *decrypted, unsigned int *encrypted, unsigned char *key)
{
    //Get thread
    const int tx = threadIdx.x + (blockIdx.x * blockDim.x);
    
    unsigned int deciphered[2];
    deciphered[0] = encrypted[0];
    deciphered[1] = encrypted[1];
    decipher(32, deciphered, (unsigned int*)key);
    
    if (tx == 0)
    {
        decrypted[0] = deciphered[0] ^ (unsigned int)1;
        decrypted[1] = deciphered[1] ^ (unsigned int)2;
    }
    
    //divide work on threads
    int i = (tx + 1) * 2;
    
    deciphered[0] = encrypted[i];
    deciphered[1] = encrypted[i+1];
    
    decipher(32, deciphered, (unsigned int*)key);
    decrypted[i] = deciphered[0] ^ encrypted[i-2];
    decrypted[i+1] = deciphered[1] ^ encrypted[i-1];
}   

__global__ void reconstruct_secret(unsigned char *result, unsigned int *decrypted)
{
    /*
    decrypted: pointer to the decrypted data
    result: pointer to where to store the unshuffled data
    */

    //Get thread
    const int tx = threadIdx.x + (blockIdx.x * blockDim.x);

    //Divide work on each thread, max 10000 threads
    if (tx < 10000)
    {
        unsigned int element = decrypted[tx];
        result[(element >> 8) % 10000] = element & 0xff;
    }

}



__device__ void decipher(unsigned int num_rounds, unsigned int v[2], unsigned int const key[4])
{
    /*
    num_rounds -- the number of iterations in the algorithm, 32 is reccomended
    input_data -- the input data to use, 32 bits of the first 2 elements are used
    key -- 128-bit key to use
    */
    unsigned int i;
    unsigned int v0=v[0], v1=v[1], delta=0x9E3779B9, sum=delta*num_rounds;

    for (i=0; i < num_rounds; i++) {
        v1 -= (((v0 << 4) ^ (v0 >> 5)) + v0) ^ (sum + key[(sum>>11) & 3]);
        sum -= delta;
        v0 -= (((v1 << 4) ^ (v1 >> 5)) + v1) ^ (sum + key[sum & 3]);
    }
    v[0]=v0; v[1]=v1;
}


